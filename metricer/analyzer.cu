#include "hip/hip_runtime.h"
#include <map>
#include <vector>
#include <string>
#include <fstream>
#include <cstdlib>
#include <algorithm>

#include "analyzer.h"
#include "analyzer_utils.h"
#include "analyzer_helper.hpp"

using namespace yosemite;

typedef struct Stats{
    uint64_t num_allocs;
    uint64_t num_kernels;
    uint64_t cur_mem_usage;
    uint64_t max_mem_usage;
    uint64_t max_mem_accesses_per_kernel;
    uint64_t avg_mem_accesses;
    uint64_t tot_mem_accesses;
    std::string max_mem_accesses_kernel;

    Stats() = default;

    ~Stats() = default;
} Stats_t;

static Stats_t _stats;

static uint64_t _timer = 0;

std::map<uint64_t, KernelEvent_t> kernel_events;
std::map<uint64_t, AllocEvent_t> alloc_events;
std::map<DevPtr, AllocEvent_t> active_memories;

std::map<std::string, uint32_t> kernel_invocations;


YosemiteResult yosemite_alloc_callback(DevPtr ptr, size_t size, int type) {
    AllocEvent_t event;
    event.addr = ptr;
    event.size = size;
    event.alloc_type = type;
    alloc_events.emplace(_timer, event);
    active_memories.emplace(ptr, event);

    _stats.num_allocs++;
    _stats.cur_mem_usage += size;
    _stats.max_mem_usage = std::max(_stats.max_mem_usage, _stats.cur_mem_usage);
    _timer++;
    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_free_callback(DevPtr ptr) {
    auto it = active_memories.find(ptr);
    if (it == active_memories.end()) {
        return YOSEMITE_ERROR;
    }
    _stats.cur_mem_usage -= it->second.size;
    active_memories.erase(it);

    _timer++;
    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_memcpy_callback() {
    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_memset_callback() {
    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_kernel_start_callback(std::string kernel_name) {
    KernelEvent_t event;
    event.kernel_name = kernel_name;
    kernel_events.emplace(_timer, event);

    if (kernel_invocations.find(kernel_name) == kernel_invocations.end()) {
        kernel_invocations.emplace(kernel_name, 1);
    } else {
        kernel_invocations[kernel_name]++;
    }

    _stats.num_kernels++;
    _timer++;
    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_kernel_end_callback(uint64_t mem_accesses) {
    KernelEvent_t& event = std::prev(kernel_events.end())->second;
    event.mem_accesses = mem_accesses;

    return YOSEMITE_SUCCESS;
}


YosemiteResult yosemite_dump_stats() {
    const char* env_filename = std::getenv("METRICS_FILE_NAME");
    std::string filename;
    if (env_filename) {
        fprintf(stdout, "METRICS_FILE_NAME: %s\n", env_filename);
        filename = std::string(env_filename) + "_" + getCurrentDateTime();
    } else {
        filename = "metrics_" + getCurrentDateTime();
        fprintf(stdout, "No filename specified. Using default filename: %s\n", filename.c_str());
    }
    filename += ".log";
    printf("Dumping traces to %s\n", filename.c_str());

    std::ofstream out(filename);
    int count = 0;
    for (auto event : alloc_events) {
        out << "Alloc(" << event.second.alloc_type << ") " << count << ":\t" << event.second.addr << " " << event.second.size << " (" << format_size(event.second.size) << ")" << std::endl;
        count++;
    }
    out << std::endl;

    count = 0;
    for (auto event : kernel_events) {
        out << "Kernel " << count << " (refs=" << event.second.mem_accesses << "):\t" << event.second.kernel_name << std::endl;
        _stats.tot_mem_accesses += event.second.mem_accesses;
        if (_stats.max_mem_accesses_per_kernel < event.second.mem_accesses) {
            _stats.max_mem_accesses_kernel = event.second.kernel_name;
            _stats.max_mem_accesses_per_kernel = event.second.mem_accesses;
        }
        count++;
    }
    out << std::endl;

    // sort kernel_invocations by number of invocations in descending order
    std::vector<std::pair<std::string, uint32_t>> sorted_kernel_invocations(kernel_invocations.begin(), kernel_invocations.end());
    std::sort(sorted_kernel_invocations.begin(), sorted_kernel_invocations.end(), [](const std::pair<std::string, uint32_t>& a, const std::pair<std::string, uint32_t>& b) {
        return a.second > b.second;
    });
    for (auto kernel : sorted_kernel_invocations) {
        out << "InvCount=" << kernel.second << "\t" << kernel.first << std::endl;
    }
    out << std::endl;

    _stats.avg_mem_accesses = _stats.tot_mem_accesses / _stats.num_kernels;
    out << "Number of allocations: " << _stats.num_allocs << std::endl;
    out << "Number of kernels: " << _stats.num_kernels << std::endl;
    out << "Maximum memory usage: " << _stats.max_mem_usage << "B (" << format_size(_stats.max_mem_usage) << ")" << std::endl;
    out << "Maximum memory accesses kernel: " << _stats.max_mem_accesses_kernel << std::endl;
    out << "Maximum memory accesses per kernel: " << _stats.max_mem_accesses_per_kernel << " (" << format_number(_stats.max_mem_accesses_per_kernel) << ")" << std::endl;
    out << "Average memory accesses per kernel: " << _stats.avg_mem_accesses << " (" << format_number(_stats.avg_mem_accesses) << ")"  << std::endl;
    out << "Total memory accesses: " << _stats.tot_mem_accesses << " (" << format_number(_stats.tot_mem_accesses) << ")"  << std::endl;
    out.close();

    return YOSEMITE_SUCCESS;
}