#include "yosemite.h"

#include "backend/nvbit_app_metric.h"


YosemiteAnalysisTool_t yosemite_tool;


/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We
 * typically do initializations in this call. In this case for instance we get
 * some environment variables values which we use as input arguments to the tool
 */
void nvbit_at_init() {
    yosemite_init(yosemite_tool);

    yosemite_tool = YOSEMITE_APP_METRICE;
    if (yosemite_tool == YOSEMITE_APP_METRICE) {
        app_metric_nvbit_at_init();
    }
}

void nvbit_tool_init(hipCtx_t ctx) {
    if (yosemite_tool == YOSEMITE_APP_METRICE) {
        app_metric_nvbit_tool_init(ctx);
    }
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    
    if (yosemite_tool == YOSEMITE_APP_METRICE) {
        app_metric_nvbit_at_cuda_event(ctx, is_exit, cbid, name, params, pStatus);
    }

}

void nvbit_at_term() {
    if (yosemite_tool == YOSEMITE_APP_METRICE) {
        app_metric_nvbit_at_term();
    }
}
